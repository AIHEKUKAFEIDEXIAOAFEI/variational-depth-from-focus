// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ### Final Project: Variational Depth from Focus
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, markus.schlaffer@in.tum.de, p070

#include <utils.cuh>
#include <iostream>
#include <openCVHelpers.h>
#include <cstring>

#include <hip/hip_runtime.h>
#include <stdio.h>

#ifndef __USE_MATH_DEFINES
#define _USE_MATH_DEFINES
#include <math.h>
#endif
#include <cstdlib>

using namespace std;
using namespace cv;

namespace vdff {
  string prev_file = "";
  int prev_line = 0;
  void cuda_check(string file, int line)
  {
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
      {
	cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
	if (prev_line>0) cout << "Previous CUDA call:" << endl << prev_file << ", line " << prev_line << endl;
	exit(1);
      }
    prev_file = file;
    prev_line = line;
  }

  namespace Utils {
    // parameter processing: template specialization for T=bool
    template<> inline bool getParam<bool>(std::string param, bool &var, int argc, char **argv)
    {
      const char *c_param = param.c_str();
      for(int i=argc-1; i>=1; i--)
	{
	  if (argv[i][0]!='-') continue;
	  if (strcmp(argv[i]+1, c_param)==0)
	    {
	      if (!(i+1<argc) || argv[i+1][0]=='-') { var = true; return true; }
	      std::stringstream ss;
	      ss << argv[i+1];
	      ss >> var;
	      return (bool)ss;
	    }
	}
      return false;
    }

    void printTiming(CUDATimer &timer, const string& launchedKernel) {
      cout << "Elapsed time";
  
      if (!launchedKernel.empty())
	cout << " for " << launchedKernel;

      cout << ": " << timer.toc() << " ms" << endl;
    }

    float getAverage(const vector<float> &v) {
      float sum = 0.0f;
      for (size_t i = 0; i < v.size(); ++i)
	sum += v[i];
  
      return sum / v.size();
    }

    hipDeviceProp_t queryDeviceProperties() {
      int nrDevices;
      hipGetDeviceCount(&nrDevices); CUDA_CHECK;

      hipDeviceProp_t bestProp;
      // check for largest constant memory
      size_t maxConstantMemory = 0;

      for(int i = 0; i < nrDevices; ++i) {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, i);

	if (prop.totalConstMem > maxConstantMemory) {
	  maxConstantMemory = prop.totalConstMem;
	  bestProp = prop;
	}
      }

      return bestProp;
    }

    char waitKey2(int delay, bool hint){
      char c;
      if(hint){
	cout << "delay="<<delay<<endl;
	if(delay < 0){
	  cout<<"[CONSOLE]: press key to continue"<<endl;
	}else if (delay == 0) {
	  cout<<"[OpenCV WINDOW]: press key to continue"<<endl;
	}else{
	  cout<<"[GENERAL]: waiting for "<< delay <<" ms"<<endl;
	}
      }
      int wait=delay;
      if(wait<0) wait*=-1;
      c=waitKey(wait);
      if(delay<0){
	std::string input;
	std::getline(std::cin,input);
	c=*input.c_str();
      }
      return c;  
    }



    string getOSSeparator() {
#ifdef _WIN32
      return "\\";
#else
      return "/";
#endif
    }

    vector<string> getAllImagesFromFolder(const char *dirname, int skipNthPicture) {
      DIR *dir = NULL;
      struct dirent *entry;
      vector<string> allImages;

      dir = opendir(dirname);

      if (!dir) {
	cerr << "Could not open directory " << dirname << ". Exiting..." << endl;
	exit(1);
      }
  
      const string sep = getOSSeparator();
      string dirStr = string(dirname);

      while(entry = readdir(dir)) {
	if (strstr(entry->d_name, ".png") ||
	    strstr(entry->d_name, ".jpg") ||
	    strstr(entry->d_name, ".tif")) {
	  string fileName(entry->d_name);
	  string fullPath = dirStr + sep + fileName;
	  allImages.push_back(fullPath);
	}
      }
      closedir(dir);

      // sort string alphabetically
      std::sort(allImages.begin(), allImages.end());

      // delete some pictures if desired
      if (skipNthPicture > 1) {

	// some sanity check
	if (skipNthPicture >= allImages.size()) {
	  cerr << "You can not skip " << skipNthPicture << " since there are only " << allImages.size() 
	       << " pictures in your chosen folder.\nPlease adjust your parameter." << endl;
	  exit(1);
	}
    
	vector<string> reduced;
	for (size_t i = 0; i < allImages.size(); ++i) {
	  if ((i % skipNthPicture) == 0)
	    continue;
      
	  reduced.push_back(allImages.at(i));
	}
	return reduced;
      }
      else {
	return allImages;
      }
    }

    void getAvailableGlobalMemory(size_t *free, size_t *total, bool print) {
      hipMemGetInfo(free, total); CUDA_CHECK;
      if(print){
	printf("AvailableGlobalMemory: %0.5f / %0.5f MB\n",*free/1e6f,*total/1e6f);
      }
    }

    void memprint() {
      size_t free,total;
      hipMemGetInfo(&free,&total); CUDA_CHECK;
      printf("AvailableGlobalMemory: %0.5f / %0.5f MB\n",free/1e6f,total/1e6f);
    }
  }
}