#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ### Final Project: Variational Depth from Focus
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, markus.schlaffer@in.tum.de, p070

#include <utils.cuh>
#include <helper.h>
#include <iostream>
#include <openCVHelpers.h>
#include <cstring>
#include <opencv2/contrib/contrib.hpp>

#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;
using namespace cv;

void printTiming(CUDATimer &timer, const string& launchedKernel) {
  cout << "Elapsed time";
  
  if (!launchedKernel.empty())
    cout << " for " << launchedKernel;

  cout << ": " << timer.toc() << " ms" << endl;
}

float getAverage(const vector<float> &v) {
  float sum = 0.0f;
  for (size_t i = 0; i < v.size(); ++i)
    sum += v[i];
  
  return sum / v.size();
}

hipDeviceProp_t queryDeviceProperties() {
  int nrDevices;
  hipGetDeviceCount(&nrDevices); CUDA_CHECK;

  hipDeviceProp_t bestProp;
  // check for largest constant memory
  size_t maxConstantMemory = 0;

  for(int i = 0; i < nrDevices; ++i) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);

    if (prop.totalConstMem > maxConstantMemory) {
      maxConstantMemory = prop.totalConstMem;
      bestProp = prop;
    }
  }

  return bestProp;
}

void imagesc(std::string title, cv::Mat mat, int x, int y) {
  double min,max;
  cv::minMaxLoc(mat,&min,&max);

  Mat scaled = mat;
  Mat meanCols;
  reduce(mat, meanCols, 0, CV_REDUCE_AVG );

  Mat mean;
  reduce(meanCols, mean, 1, CV_REDUCE_AVG);
    
  cout << "Max value: " << max << endl;
  cout << "Mean value: " << mean.at<float>(0) << endl;
  cout << "Min value: " << min << endl;
    
  if (std::abs(max) > 0.0000001f)
    scaled /= max;

  showImage(title, scaled, x,y);
}

char waitKey2(int delay, bool hint){
  char c;
  if(hint){
    cout << "delay="<<delay<<endl;
    if(delay < 0){
      cout<<"[CONSOLE]: press key to continue"<<endl;
    }else if (delay == 0) {
      cout<<"[OpenCV WINDOW]: press key to continue"<<endl;
    }else{
      cout<<"[GENERAL]: waiting for "<< delay <<" ms"<<endl;
    }
  }
  int wait=delay;
  if(wait<0) wait*=-1;
  c=waitKey(wait);
  if(delay<0){
    std::string input;
    std::getline(std::cin,input);
    c=*input.c_str();
  }
  return c;  
}

void createOptimallyPaddedImageForDCT(const Mat& img, Mat& paddedImg, 
				      int &paddingX, int &paddingY) {
  // pad init if it is not divisible by 2
  int maxVecSize = max(img.rows, img.cols);
  int optVecSize = getOptimalDFTSize((maxVecSize+1)/2)*2;

  paddingX = optVecSize - img.cols;
  paddingY = optVecSize - img.rows;

  int top, bottom, left, right;
  top = bottom = paddingY / 2;
  left = right = paddingX / 2;
  bottom += paddingY % 2 == 1;  
  right += paddingX % 2 == 1;

  if (paddingX == 0 && paddingY == 0) {
    paddedImg = img.clone();
  }
  else {
    copyMakeBorder(img, paddedImg, top, bottom, left, right, BORDER_CONSTANT, Scalar(0));
  }  
}

void showDepthImage(const string &wndTitle, const Mat& img, int posX, int posY, bool doResize) {
  double min, max;
  minMaxIdx(img, &min, &max);

  Mat depthMap;
  float scale = 255.0f / (max - min);
  img.convertTo(depthMap, CV_8UC1, scale, -min*scale);

  Mat heatMap;
  applyColorMap(depthMap, heatMap, cv::COLORMAP_JET);

  if (doResize)
    resize(heatMap, heatMap, Size(), 0.5, 0.5);
  
  showImage(wndTitle, heatMap, posX, posY);
}

string getOSSeparator() {
#ifdef _WIN32
  return "\\";
#else
  return "/";
#endif
}

vector<string> getAllImagesFromFolder(const char *dirname) {
  DIR *dir = NULL;
  struct dirent *entry;
  vector<string> allImages;

  dir = opendir(dirname);

  if (!dir) {
    cerr << "Could not open directory " << dirname << ". Exiting..." << endl;
    exit(1);
  }
  
  const string sep = getOSSeparator();
  string dirStr = string(dirname);

  while(entry = readdir(dir)) {
    if (strstr(entry->d_name, ".png") ||
	strstr(entry->d_name, ".jpg") ||
	strstr(entry->d_name, ".tif")) {
      string fileName(entry->d_name);
      string fullPath = dirStr + sep + fileName;
      allImages.push_back(fullPath);
    }
  }
  closedir(dir);

  // sort string alphabetically
  std::sort(allImages.begin(), allImages.end());
  return allImages;
}

void getAvailableGlobalMemory(size_t *free, size_t *total, bool print) {
  hipMemGetInfo(free, total); CUDA_CHECK;
  if(print){
    printf("AvailableGlobalMemory: %0.5f / %0.5f MB\n",*free/1e6f,*total/1e6f);
  }
}

void memprint() {
  size_t free,total;
  hipMemGetInfo(&free,&total); CUDA_CHECK;
  printf("AvailableGlobalMemory: %0.5f / %0.5f MB\n",free/1e6f,total/1e6f);
}

void printSharpnessValues(float *l_sharpness, size_t x, size_t y, size_t w, size_t h, size_t n) {
  cout << "Sharpness values at (y: " << y << ", x: " << x << "): ";
  for(size_t i = 0; i < n; ++i) {
    cout << l_sharpness[x + y*w + i*w*h] << ", ";
  }
  cout << endl;
}

// degree is polynomial degree!
void printCoefficients(float *l_coef, size_t x, size_t y, size_t w, size_t h, size_t degree) {
  cout << "Coefficient at (y: " << y << ", x: " << x << "): ";
  for(size_t i = 0; i < degree+1; ++i) {
    cout << l_coef[x + y*w + i*w*h] << ", ";
  }
  cout << endl;
}

// degree is polynomial degree! (not degree of the Derivatives.)
void printDerivativeCoefficients(float *l_coefDeriv, size_t x, size_t y, size_t w, size_t h, size_t degree) {
  cout << "Derivative Coefficient at (y: " << y << ", x: " << x << "): ";
  for(size_t i = 0; i < degree; ++i) {
    cout << l_coefDeriv[x + y*w + i*w*h] << ", ";    
  }
  cout << endl;
}